
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.51
Copyright (C) 2018  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA

#include <cuda.h>
#include "CalculateForceCUDAKernel.cuh"
#include "CalculateEnergyCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "cub/cub.cuh"
#include <stdio.h>
#define NUMBER_OF_NEIGHBOR_CELL 27

using namespace cub;

void CallBoxInterForceGPU(VariablesCUDA *vars,
                          vector<int> &cellVector,
                          vector<int> &cellStartIndex,
                          std::vector<std::vector<int> > &neighborList,
                          vector<int> &mapParticleToCell,
                          XYZArray const &currentCoords,
                          XYZArray const &currentCOM,
                          BoxDimensions const &boxAxes,
                          bool electrostatic,
                          vector<double> &particleCharge,
                          vector<int> &particleKind,
                          vector<int> &particleMol,
                          double &rT11,
                          double &rT12,
                          double &rT13,
                          double &rT22,
                          double &rT23,
                          double &rT33,
                          double &vT11,
                          double &vT12,
                          double &vT13,
                          double &vT22,
                          double &vT23,
                          double &vT33,
                          bool sc_coul,
                          double sc_sigma_6,
                          double sc_alpha,
                          uint sc_power,
                          uint const box)
{
  int atomNumber = currentCoords.Count();
  int molNumber = currentCOM.Count();
  int neighborListCount = neighborList.size() * NUMBER_OF_NEIGHBOR_CELL;
  int numberOfCells = neighborList.size();
  int *gpu_particleKind;
  int *gpu_particleMol;
  int *gpu_neighborList, *gpu_cellStartIndex;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(numberOfCells * NUMBER_OF_NEIGHBOR_CELL);

  // Convert neighbor list to 1D array
  std::vector<int> neighborlist1D(neighborListCount);
  for(int i=0; i<neighborList.size(); i++) {
    for(int j=0; j<NUMBER_OF_NEIGHBOR_CELL; j++) {
      neighborlist1D[i*NUMBER_OF_NEIGHBOR_CELL + j] = neighborList[i][j];
    }
  }

  cudaMalloc((void**) &gpu_neighborList, neighborListCount * sizeof(int));
  cudaMalloc((void**) &gpu_cellStartIndex,
      cellStartIndex.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleCharge,
      particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));
  cudaMalloc(&vars->gpu_rT11, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT12, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT13, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT22, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT23, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT33, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT11, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT12, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT13, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT22, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT23, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT33, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
      threadsPerBlock * sizeof(double));

  cudaMemcpy(vars->gpu_mapParticleToCell, &mapParticleToCell[0],
      atomNumber * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_neighborList, &neighborlist1D[0],
      neighborListCount * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_cellStartIndex, &cellStartIndex[0],
      cellStartIndex.size() * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_cellVector, &cellVector[0],
      atomNumber * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comx, currentCOM.x, molNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comy, currentCOM.y, molNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comz, currentCOM.z, molNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
      particleCharge.size() * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
      particleKind.size() * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0],
      particleMol.size() * sizeof(int),
      cudaMemcpyHostToDevice);

  BoxInterForceGPU <<< blocksPerGrid, threadsPerBlock>>>(gpu_cellStartIndex,
      vars->gpu_cellVector,
      gpu_neighborList,
      numberOfCells,
      atomNumber,
      vars->gpu_mapParticleToCell,
      vars->gpu_x,
      vars->gpu_y,
      vars->gpu_z,
      vars->gpu_comx,
      vars->gpu_comy,
      vars->gpu_comz,
      boxAxes.GetAxis(box).x,
      boxAxes.GetAxis(box).y,
      boxAxes.GetAxis(box).z,
      electrostatic,
      gpu_particleCharge,
      gpu_particleKind,
      gpu_particleMol,
      vars->gpu_rT11,
      vars->gpu_rT12,
      vars->gpu_rT13,
      vars->gpu_rT22,
      vars->gpu_rT23,
      vars->gpu_rT33,
      vars->gpu_vT11,
      vars->gpu_vT12,
      vars->gpu_vT13,
      vars->gpu_vT22,
      vars->gpu_vT23,
      vars->gpu_vT33,
      vars->gpu_sigmaSq,
      vars->gpu_epsilon_Cn,
      vars->gpu_n,
      vars->gpu_VDW_Kind,
      vars->gpu_isMartini,
      vars->gpu_count,
      vars->gpu_rCut,
      vars->gpu_rCutCoulomb,
      vars->gpu_rCutLow,
      vars->gpu_rOn,
      vars->gpu_alpha,
      vars->gpu_ewald,
      vars->gpu_diElectric_1,
      vars->gpu_cell_x[box],
      vars->gpu_cell_y[box],
      vars->gpu_cell_z[box],
      vars->gpu_Invcell_x[box],
      vars->gpu_Invcell_y[box],
      vars->gpu_Invcell_z[box],
      vars->gpu_nonOrth,
      sc_coul,
      sc_sigma_6,
      sc_alpha,
      sc_power,
      vars->gpu_rMin,
      vars->gpu_rMaxSq,
      vars->gpu_expConst,
      vars->gpu_molIndex,
      vars->gpu_kindIndex,
      vars->gpu_lambdaVDW,
      vars->gpu_lambdaCoulomb,
      vars->gpu_isFraction,
      box);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaDeviceSynchronize();
  // ReduceSum // Virial of LJ
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
      gpu_final_value, pair1.size());
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
      gpu_final_value, pair1.size());
  cudaMemcpy(&vT11, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT12,
      gpu_final_value, pair1.size());
  cudaMemcpy(&vT12, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT13,
      gpu_final_value, pair1.size());
  cudaMemcpy(&vT13, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT22,
      gpu_final_value, pair1.size());
  cudaMemcpy(&vT22, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT23,
      gpu_final_value, pair1.size());
  cudaMemcpy(&vT23, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT33,
      gpu_final_value, pair1.size());
  cudaMemcpy(&vT33, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);

  if(electrostatic) {
    // ReduceSum // Virial of Coulomb
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
        gpu_final_value, pair1.size());
    cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
        cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
        gpu_final_value, pair1.size());
    cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
        cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
        gpu_final_value, pair1.size());
    cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
        cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
        gpu_final_value, pair1.size());
    cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
        cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
        gpu_final_value, pair1.size());
    cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
        cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
        gpu_final_value, pair1.size());
    cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
        cudaMemcpyDeviceToHost);
  }

  cudaFree(vars->gpu_rT11);
  cudaFree(vars->gpu_rT12);
  cudaFree(vars->gpu_rT13);
  cudaFree(vars->gpu_rT22);
  cudaFree(vars->gpu_rT23);
  cudaFree(vars->gpu_rT33);
  cudaFree(vars->gpu_vT11);
  cudaFree(vars->gpu_vT12);
  cudaFree(vars->gpu_vT13);
  cudaFree(vars->gpu_vT22);
  cudaFree(vars->gpu_vT23);
  cudaFree(vars->gpu_vT33);
  cudaFree(d_temp_storage);
  cudaFree(gpu_pair1);
  cudaFree(gpu_pair2);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
}

void CallBoxForceGPU(VariablesCUDA *vars,
    vector<uint> pair1,
    vector<uint> pair2,
    XYZArray const &coords,
    BoxDimensions const &boxAxes,
    bool electrostatic,
    vector<double> particleCharge,
    vector<int> particleKind,
    vector<int> particleMol,
    double &REn,
    double &LJEn,
    double *aForcex,
    double *aForcey,
    double *aForcez,
    double *mForcex,
    double *mForcey,
    double *mForcez,
    int atomCount,
    int molCount,
    bool reset_force,
    bool copy_back,
    bool sc_coul,
    double sc_sigma_6,
    double sc_alpha,
    uint sc_power,
    uint const box)
{
  int atomNumber = coords.Count();
  int *gpu_pair1, *gpu_pair2, *gpu_particleKind, *gpu_particleMol;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_REn, *gpu_LJEn;
  double *gpu_final_REn, *gpu_final_LJEn;
  double cpu_final_REn, cpu_final_LJEn;

  if(reset_force) {
    cudaMemset(vars->gpu_aForcex, 0, atomCount * sizeof(double));
    cudaMemset(vars->gpu_aForcey, 0, atomCount * sizeof(double));
    cudaMemset(vars->gpu_aForcez, 0, atomCount * sizeof(double));
    cudaMemset(vars->gpu_mForcex, 0, molCount * sizeof(double));
    cudaMemset(vars->gpu_mForcey, 0, molCount * sizeof(double));
    cudaMemset(vars->gpu_mForcez, 0, molCount * sizeof(double));
  }

  cudaMalloc((void**) &gpu_pair1, pair1.size() * sizeof(int));
  cudaMalloc((void**) &gpu_pair2, pair2.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleCharge,
      particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_REn, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_LJEn, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_final_REn, sizeof(double));
  cudaMalloc((void**) &gpu_final_LJEn, sizeof(double));

  // Copy necessary data to GPU
  cudaMemcpy(gpu_pair1, &pair1[0], pair1.size() * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_pair2, &pair2[0], pair2.size() * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
      particleCharge.size() * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
      particleKind.size() * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], particleMol.size() * sizeof(int),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(pair1.size() / threadsPerBlock) + 1;
  BoxForceGPU <<< blocksPerGrid, threadsPerBlock>>>(gpu_pair1,
      gpu_pair2,
      vars->gpu_x,
      vars->gpu_y,
      vars->gpu_z,
      boxAxes.GetAxis(box).x,
      boxAxes.GetAxis(box).y,
      boxAxes.GetAxis(box).z,
      electrostatic,
      gpu_particleCharge,
      gpu_particleKind,
      gpu_particleMol,
      gpu_REn,
      gpu_LJEn,
      pair1.size(),
      vars->gpu_sigmaSq,
      vars->gpu_epsilon_Cn,
      vars->gpu_n,
      vars->gpu_VDW_Kind,
      vars->gpu_isMartini,
      vars->gpu_count,
      vars->gpu_rCut,
      vars->gpu_rCutCoulomb,
      vars->gpu_rCutLow,
      vars->gpu_rOn,
      vars->gpu_alpha,
      vars->gpu_ewald,
      vars->gpu_diElectric_1,
      vars->gpu_nonOrth,
      vars->gpu_cell_x[box],
      vars->gpu_cell_y[box],
      vars->gpu_cell_z[box],
      vars->gpu_Invcell_x[box],
      vars->gpu_Invcell_y[box],
      vars->gpu_Invcell_z[box],
      vars->gpu_aForcex,
      vars->gpu_aForcey,
      vars->gpu_aForcez,
      vars->gpu_mForcex,
      vars->gpu_mForcey,
      vars->gpu_mForcez,
      sc_coul,
      sc_sigma_6,
      sc_alpha,
      sc_power,
      vars->gpu_rMin,
      vars->gpu_rMaxSq,
      vars->gpu_expConst,
      vars->gpu_molIndex,
      vars->gpu_kindIndex,
      vars->gpu_lambdaVDW,
      vars->gpu_lambdaCoulomb,
      vars->gpu_isFraction,
      box);

      checkLastErrorCUDA(__FILE__, __LINE__);
      // ReduceSum
      void * d_temp_storage = NULL;
      size_t temp_storage_bytes = 0;
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
          gpu_final_REn, pair1.size());
      CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
          gpu_final_REn, pair1.size());
      cudaFree(d_temp_storage);

      // LJ ReduceSum
      d_temp_storage = NULL;
      temp_storage_bytes = 0;
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
          gpu_final_LJEn, pair1.size());
      CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
          gpu_final_LJEn, pair1.size());
      cudaFree(d_temp_storage);
      // Copy back the result to CPU ! :)
      CubDebugExit(cudaMemcpy(&cpu_final_REn, gpu_final_REn, sizeof(double),
            cudaMemcpyDeviceToHost));
      CubDebugExit(cudaMemcpy(&cpu_final_LJEn, gpu_final_LJEn, sizeof(double),
            cudaMemcpyDeviceToHost));
      REn = cpu_final_REn;
      LJEn = cpu_final_LJEn;

      if(copy_back) {
        CubDebugExit(cudaMemcpy(aForcex, vars->gpu_aForcex,
              sizeof(double) * atomCount,
              cudaMemcpyDeviceToHost));
        CubDebugExit(cudaMemcpy(aForcey, vars->gpu_aForcey,
              sizeof(double) * atomCount,
              cudaMemcpyDeviceToHost));
        CubDebugExit(cudaMemcpy(aForcez, vars->gpu_aForcez,
              sizeof(double) * atomCount,
              cudaMemcpyDeviceToHost));
        CubDebugExit(cudaMemcpy(mForcex, vars->gpu_mForcex,
              sizeof(double) * molCount,
              cudaMemcpyDeviceToHost));
        CubDebugExit(cudaMemcpy(mForcey, vars->gpu_mForcey,
              sizeof(double) * molCount,
              cudaMemcpyDeviceToHost));
        CubDebugExit(cudaMemcpy(mForcez, vars->gpu_mForcez,
              sizeof(double) * molCount,
              cudaMemcpyDeviceToHost));
      }
      cudaDeviceSynchronize();

      cudaFree(gpu_pair1);
      cudaFree(gpu_pair2);
      cudaFree(gpu_particleCharge);
      cudaFree(gpu_particleKind);
      cudaFree(gpu_particleMol);
      cudaFree(gpu_REn);
      cudaFree(gpu_LJEn);
      cudaFree(gpu_final_REn);
      cudaFree(gpu_final_LJEn);
}

void CallVirialReciprocalGPU(VariablesCUDA *vars,
    XYZArray const &currentCoords,
    XYZArray const &currentCOMDiff,
    vector<double> &particleCharge,
    double &rT11,
    double &rT12,
    double &rT13,
    double &rT22,
    double &rT23,
    double &rT33,
    uint imageSize,
    double constVal,
    uint box)
{
  int atomNumber = currentCoords.Count();
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  cudaMalloc((void**) &gpu_particleCharge,
      particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));

  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dx, currentCOMDiff.x, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dy, currentCOMDiff.y, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dz, currentCOMDiff.z, atomNumber * sizeof(double),
      cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
      particleCharge.size() * sizeof(double),
      cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  VirialReciprocalGPU <<< blocksPerGrid,
                      threadsPerBlock>>>(vars->gpu_x,
                          vars->gpu_y,
                          vars->gpu_z,
                          vars->gpu_dx,
                          vars->gpu_dy,
                          vars->gpu_dz,
                          vars->gpu_kxRef[box],
                          vars->gpu_kyRef[box],
                          vars->gpu_kzRef[box],
                          vars->gpu_prefactRef[box],
                          vars->gpu_hsqrRef[box],
                          vars->gpu_sumRref[box],
                          vars->gpu_sumIref[box],
                          gpu_particleCharge,
                          vars->gpu_rT11,
                          vars->gpu_rT12,
                          vars->gpu_rT13,
                          vars->gpu_rT22,
                          vars->gpu_rT23,
                          vars->gpu_rT33,
                          constVal,
                          imageSize,
                          atomNumber);
  checkLastErrorCUDA(__FILE__, __LINE__);
  // ReduceSum // Virial of Reciprocal
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
      gpu_final_value, imageSize);
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
      gpu_final_value, imageSize);
  cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
      gpu_final_value, imageSize);
  cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
      gpu_final_value, imageSize);
  cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
      gpu_final_value, imageSize);
  cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
      gpu_final_value, imageSize);
  cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
      gpu_final_value, imageSize);
  cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
      cudaMemcpyDeviceToHost);

  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
  cudaFree(d_temp_storage);
}

__global__ void BoxInterForceGPU(int *gpu_pair1,
    int *gpu_pair2,
    double *gpu_x,
    double *gpu_y,
    double *gpu_z,
    double *gpu_comx,
    double *gpu_comy,
    double *gpu_comz,
    double xAxes,
    double yAxes,
    double zAxes,
    bool electrostatic,
    double *gpu_particleCharge,
    int *gpu_particleKind,
    int *gpu_particleMol,
    double *gpu_rT11,
    double *gpu_rT12,
    double *gpu_rT13,
    double *gpu_rT22,
    double *gpu_rT23,
    double *gpu_rT33,
    double *gpu_vT11,
    double *gpu_vT12,
    double *gpu_vT13,
    double *gpu_vT22,
    double *gpu_vT23,
    double *gpu_vT33,
    int pairSize,
    double *gpu_sigmaSq,
    double *gpu_epsilon_Cn,
    double *gpu_n,
    int *gpu_VDW_Kind,
    int *gpu_isMartini,
    int *gpu_count,
    double *gpu_rCut,
    double *gpu_rCutCoulomb,
    double *gpu_rCutLow,
    double *gpu_rOn,
    double *gpu_alpha,
    int *gpu_ewald,
    double *gpu_diElectric_1,
    double *gpu_cell_x,
    double *gpu_cell_y,
    double *gpu_cell_z,
    double *gpu_Invcell_x,
    double *gpu_Invcell_y,
    double *gpu_Invcell_z,
    int *gpu_nonOrth,
    bool sc_coul,
    double sc_sigma_6,
    double sc_alpha,
    uint sc_power,
    double *gpu_rMin,
    double *gpu_rMaxSq,
    double *gpu_expConst,
    int *gpu_molIndex,
    int *gpu_kindIndex,
    double *gpu_lambdaVDW,
    double *gpu_lambdaCoulomb,
    bool *gpu_isFraction,
    int box)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= pairSize)
    return;

  double distSq;
  double virX, virY, virZ;
  double pRF = 0.0, qi_qj, pVF = 0.0;
  double lambdaVDW = 0.0, lambdaCoulomb = 0.0;
  //tensors for VDW and real part of electrostatic
  gpu_vT11[threadID] = 0.0, gpu_vT22[threadID] = 0.0, gpu_vT33[threadID] = 0.0;
  gpu_rT11[threadID] = 0.0, gpu_rT22[threadID] = 0.0, gpu_rT33[threadID] = 0.0;
  // extra tensors reserved for later on
  gpu_vT12[threadID] = 0.0, gpu_vT13[threadID] = 0.0, gpu_vT23[threadID] = 0.0;
  gpu_rT12[threadID] = 0.0, gpu_rT13[threadID] = 0.0, gpu_rT23[threadID] = 0.0;
  double diff_comx, diff_comy, diff_comz;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[gpu_pair1[threadID]],
        gpu_y[gpu_pair1[threadID]], gpu_z[gpu_pair1[threadID]],
        gpu_x[gpu_pair2[threadID]], gpu_y[gpu_pair2[threadID]],
        gpu_z[gpu_pair2[threadID]], xAxes, yAxes, zAxes, xAxes / 2.0,
        yAxes / 2.0, zAxes / 2.0, cutoff, gpu_nonOrth[0],
        gpu_cell_x, gpu_cell_y, gpu_cell_z, gpu_Invcell_x, gpu_Invcell_y,
        gpu_Invcell_z)) {
    int cA = gpu_particleCharge[gpu_pair1[threadID]];
    int cB = gpu_particleCharge[gpu_pair2[threadID]];
    int kA = gpu_particleKind[gpu_pair1[threadID]];
    int kB = gpu_particleKind[gpu_pair2[threadID]];
    int mA = gpu_particleMol[gpu_pair1[threadID]];
    int mB = gpu_particleMol[gpu_pair2[threadID]];

    lambdaVDW = DeviceGetLambdaVDW(mA, kA, mB, kB, box, gpu_isFraction,
        gpu_molIndex, gpu_kindIndex, gpu_lambdaVDW);

    diff_comx = gpu_comx[mA] - gpu_comx[mB];
    diff_comy = gpu_comy[mA] - gpu_comy[mB];
    diff_comz = gpu_comz[mA] - gpu_comz[mB];

    diff_comx = MinImageSignedGPU(diff_comx, xAxes, xAxes / 2.0);
    diff_comy = MinImageSignedGPU(diff_comy, yAxes, yAxes / 2.0);
    diff_comz = MinImageSignedGPU(diff_comz, zAxes, zAxes / 2.0);

    if(electrostatic) {
      qi_qj = cA * cB;
      lambdaCoulomb = DeviceGetLambdaCoulomb(mA, kA, mB, kB, box,
          gpu_isFraction, gpu_molIndex,
          gpu_kindIndex, gpu_lambdaCoulomb);
      pRF = CalcCoulombForceGPU(distSq, qi_qj, gpu_VDW_Kind[0], gpu_ewald[0],
          gpu_isMartini[0], gpu_alpha[box],
          gpu_rCutCoulomb[box], gpu_diElectric_1[0],
          gpu_sigmaSq, sc_coul, sc_sigma_6, sc_alpha,
          sc_power, lambdaCoulomb, gpu_count[0],
          kA, kB);

      gpu_rT11[threadID] = pRF * (virX * diff_comx);
      gpu_rT22[threadID] = pRF * (virY * diff_comy);
      gpu_rT33[threadID] = pRF * (virZ * diff_comz);

      //extra tensor calculations
      gpu_rT12[threadID] = pRF * (0.5 * (virX * diff_comy + virY * diff_comx));
      gpu_rT13[threadID] = pRF * (0.5 * (virX * diff_comz + virZ * diff_comx));
      gpu_rT23[threadID] = pRF * (0.5 * (virY * diff_comz + virZ * diff_comy));
    }

    pVF = CalcEnForceGPU(distSq, kA, kB,
        gpu_sigmaSq, gpu_n, gpu_epsilon_Cn, gpu_rCut[0],
        gpu_rOn[0], gpu_isMartini[0], gpu_VDW_Kind[0],
        gpu_count[0], lambdaVDW, sc_sigma_6, sc_alpha,
        sc_power, gpu_rMin, gpu_rMaxSq, gpu_expConst);

    gpu_vT11[threadID] = pVF * (virX * diff_comx);
    gpu_vT22[threadID] = pVF * (virY * diff_comy);
    gpu_vT33[threadID] = pVF * (virZ * diff_comz);

    //extra tensor calculations
    gpu_vT12[threadID] = pVF * (0.5 * (virX * diff_comy + virY * diff_comx));
    gpu_vT13[threadID] = pVF * (0.5 * (virX * diff_comz + virZ * diff_comx));
    gpu_vT23[threadID] = pVF * (0.5 * (virY * diff_comz + virZ * diff_comy));
  }
}

__global__ void BoxForceGPU(int *gpu_pair1,
    int *gpu_pair2,
    double *gpu_x,
    double *gpu_y,
    double *gpu_z,
    double xAxes,
    double yAxes,
    double zAxes,
    bool electrostatic,
    double *gpu_particleCharge,
    int *gpu_particleKind,
    int *gpu_particleMol,
    double *gpu_REn,
    double *gpu_LJEn,
    int pairSize,
    double *gpu_sigmaSq,
    double *gpu_epsilon_Cn,
    double *gpu_n,
    int *gpu_VDW_Kind,
    int *gpu_isMartini,
    int *gpu_count,
    double *gpu_rCut,
    double *gpu_rCutCoulomb,
    double *gpu_rCutLow,
    double *gpu_rOn,
    double *gpu_alpha,
    int *gpu_ewald,
    double *gpu_diElectric_1,
    int *gpu_nonOrth,
    double *gpu_cell_x,
    double *gpu_cell_y,
    double *gpu_cell_z,
    double *gpu_Invcell_x,
    double *gpu_Invcell_y,
    double *gpu_Invcell_z,
    double *gpu_aForcex,
    double *gpu_aForcey,
    double *gpu_aForcez,
    double *gpu_mForcex,
    double *gpu_mForcey,
    double *gpu_mForcez,
    bool sc_coul,
    double sc_sigma_6,
    double sc_alpha,
    uint sc_power,
    double *gpu_rMin,
    double *gpu_rMaxSq,
    double *gpu_expConst,
    int *gpu_molIndex,
    int *gpu_kindIndex,
    double *gpu_lambdaVDW,
    double *gpu_lambdaCoulomb,
    bool *gpu_isFraction,
    int box)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= pairSize)
    return;
  double distSq;
  double qi_qj_fact;
  double qqFact = 167000.0;
  double virX = 0.0, virY = 0.0, virZ = 0.0;
  double forceRealx = 0.0, forceRealy = 0.0, forceRealz = 0.0;
  double forceLJx = 0.0, forceLJy = 0.0, forceLJz = 0.0;
  double lambdaVDW = 0.0, lambdaCoulomb = 0.0;
  gpu_REn[threadID] = 0.0;
  gpu_LJEn[threadID] = 0.0;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);
  if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[gpu_pair1[threadID]],
        gpu_y[gpu_pair1[threadID]], gpu_z[gpu_pair1[threadID]],
        gpu_x[gpu_pair2[threadID]], gpu_y[gpu_pair2[threadID]],
        gpu_z[gpu_pair2[threadID]], xAxes, yAxes, zAxes, xAxes / 2.0,
        yAxes / 2.0, zAxes / 2.0, cutoff, gpu_nonOrth[0], gpu_cell_x,
        gpu_cell_y, gpu_cell_z, gpu_Invcell_x, gpu_Invcell_y,
        gpu_Invcell_z)) {
    int cA = gpu_particleCharge[gpu_pair1[threadID]];
    int cB = gpu_particleCharge[gpu_pair2[threadID]];
    int kA = gpu_particleKind[gpu_pair1[threadID]];
    int kB = gpu_particleKind[gpu_pair2[threadID]];
    int mA = gpu_particleMol[gpu_pair1[threadID]];
    int mB = gpu_particleMol[gpu_pair2[threadID]];

    lambdaVDW = DeviceGetLambdaVDW(mA, kA, mB, kB, box, gpu_isFraction,
        gpu_molIndex, gpu_kindIndex, gpu_lambdaVDW);

    if(electrostatic) {
      qi_qj_fact = cA * cB * qqFact;
      lambdaCoulomb = DeviceGetLambdaCoulomb(mA, kA, mB, kB, box,
          gpu_isFraction, gpu_molIndex,
          gpu_kindIndex, gpu_lambdaCoulomb);
      gpu_REn[threadID] = CalcCoulombGPU(distSq,
          kA,
          kB,
          qi_qj_fact, gpu_rCutLow[0],
          gpu_ewald[0], gpu_VDW_Kind[0],
          gpu_alpha[box],
          gpu_rCutCoulomb[box],
          gpu_isMartini[0],
          gpu_diElectric_1[0],
          lambdaCoulomb,
          sc_coul,
          sc_sigma_6,
          sc_alpha,
          sc_power,
          gpu_sigmaSq[threadID],
          gpu_count[0]);
    }
    gpu_LJEn[threadID] = CalcEnGPU(distSq,
        kA,
        kB,
        gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
        gpu_VDW_Kind[0], gpu_isMartini[0],
        gpu_rCut[0], gpu_rOn[0], gpu_count[0],
        lambdaVDW,
        sc_sigma_6, sc_alpha, sc_power, gpu_rMin,
        gpu_rMaxSq, gpu_expConst);
    if(electrostatic) {
      double coulombVir = CalcCoulombForceGPU(distSq, qi_qj_fact,
          gpu_VDW_Kind[0], gpu_ewald[0],
          gpu_isMartini[0],
          gpu_alpha[box],
          gpu_rCutCoulomb[box],
          gpu_diElectric_1[0],
          gpu_sigmaSq, sc_coul, sc_sigma_6,
          sc_alpha, sc_power,
          lambdaCoulomb,
          gpu_count[0],
          kA,
          kB);
      forceRealx = virX * coulombVir;
      forceRealy = virY * coulombVir;
      forceRealz = virZ * coulombVir;
    }
    double pVF = CalcEnForceGPU(distSq, kA,
        kB,
        gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
        gpu_rCut[0], gpu_rOn[0], gpu_isMartini[0],
        gpu_VDW_Kind[0], gpu_count[0],
        lambdaVDW, sc_sigma_6, sc_alpha,
        sc_power, gpu_rMin, gpu_rMaxSq, gpu_expConst);
    forceLJx = virX * pVF;
    forceLJy = virY * pVF;
    forceLJz = virZ * pVF;

    atomicAdd(&gpu_aForcex[gpu_pair1[threadID]], forceRealx + forceLJx);
    atomicAdd(&gpu_aForcey[gpu_pair1[threadID]], forceRealy + forceLJy);
    atomicAdd(&gpu_aForcez[gpu_pair1[threadID]], forceRealz + forceLJz);
    atomicAdd(&gpu_aForcex[gpu_pair2[threadID]], -1.0 * (forceRealx + forceLJx));
    atomicAdd(&gpu_aForcey[gpu_pair2[threadID]], -1.0 * (forceRealy + forceLJy));
    atomicAdd(&gpu_aForcez[gpu_pair2[threadID]], -1.0 * (forceRealz + forceLJz));

    atomicAdd(&gpu_mForcex[mA], forceRealx + forceLJx);
    atomicAdd(&gpu_mForcey[mA], forceRealy + forceLJy);
    atomicAdd(&gpu_mForcez[mA], forceRealz + forceLJz);
    atomicAdd(&gpu_mForcex[mB], -1.0 * (forceRealx + forceLJx));
    atomicAdd(&gpu_mForcey[mB], -1.0 * (forceRealy + forceLJy));
    atomicAdd(&gpu_mForcez[mB], -1.0 * (forceRealz + forceLJz));
  }
}

__global__ void VirialReciprocalGPU(double *gpu_x,
    double *gpu_y,
    double *gpu_z,
    double *gpu_comDx,
    double *gpu_comDy,
    double *gpu_comDz,
    double *gpu_kxRef,
    double *gpu_kyRef,
    double *gpu_kzRef,
    double *gpu_prefactRef,
    double *gpu_hsqrRef,
    double *gpu_sumRref,
    double *gpu_sumIref,
    double *gpu_particleCharge,
    double *gpu_rT11,
    double *gpu_rT12,
    double *gpu_rT13,
    double *gpu_rT22,
    double *gpu_rT23,
    double *gpu_rT33,
    double constVal,
    uint imageSize,
    uint atomNumber)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  double factor, arg;
  int i;
  factor = gpu_prefactRef[threadID] * (gpu_sumRref[threadID] *
      gpu_sumRref[threadID] +
      gpu_sumIref[threadID] *
      gpu_sumIref[threadID]);
  gpu_rT11[threadID] = factor * (1.0 - 2.0 *
      (constVal + 1.0 / gpu_hsqrRef[threadID]) *
      gpu_kxRef[threadID] * gpu_kxRef[threadID]);
  gpu_rT12[threadID] = factor * (-2.0 *
      (constVal + 1.0 / gpu_hsqrRef[threadID]) *
      gpu_kxRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT13[threadID] = factor * (-2.0 *
      (constVal + 1.0 / gpu_hsqrRef[threadID]) *
      gpu_kxRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT22[threadID] = factor * (1.0 - 2.0 *
      (constVal + 1.0 / gpu_hsqrRef[threadID]) *
      gpu_kyRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT23[threadID] = factor * (-2.0 *
      (constVal + 1.0 / gpu_hsqrRef[threadID]) *
      gpu_kyRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT33[threadID] = factor * (1.0 - 2.0 *
      (constVal + 1.0 / gpu_hsqrRef[threadID]) *
      gpu_kzRef[threadID] * gpu_kzRef[threadID]);

  //Intramolecular part
  for(i = 0; i < atomNumber; i++) {
    arg = DotProductGPU(gpu_kxRef[threadID], gpu_kyRef[threadID],
        gpu_kzRef[threadID], gpu_x[i], gpu_y[i], gpu_z[i]);

    factor = gpu_prefactRef[threadID] * 2.0 *
      (gpu_sumIref[threadID] * cos(arg) - gpu_sumRref[threadID] * sin(arg)) *
      gpu_particleCharge[i];

    gpu_rT11[threadID] += factor * (gpu_kxRef[threadID] * gpu_comDx[i]);
    gpu_rT12[threadID] += factor * 0.5 * (gpu_kxRef[threadID] * gpu_comDy[i] +
        gpu_kyRef[threadID] * gpu_comDx[i]);
    gpu_rT13[threadID] += factor * 0.5 * (gpu_kxRef[threadID] * gpu_comDz[i] +
        gpu_kzRef[threadID] * gpu_comDx[i]);
    gpu_rT22[threadID] += factor * (gpu_kyRef[threadID] * gpu_comDy[i]);
    gpu_rT13[threadID] += factor * 0.5 * (gpu_kyRef[threadID] * gpu_comDz[i] +
        gpu_kzRef[threadID] * gpu_comDy[i]);
    gpu_rT33[threadID] += factor * (gpu_kzRef[threadID] * gpu_comDz[i]);
  }
}

__device__ double CalcEnForceGPU(double distSq, int kind1, int kind2,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn, double gpu_rCut,
    double gpu_rOn, int gpu_isMartini,
    int gpu_VDW_Kind, int gpu_count,
    double gpu_lambdaVDW, double sc_sigma_6,
    double sc_alpha, uint sc_power,
    double *gpu_rMin, double *gpu_rMaxSq,
    double *gpu_expConst)
{
  if((gpu_rCut * gpu_rCut) < distSq) {
    return 0.0;
  }

  int index = FlatIndexGPU(kind1, kind2, gpu_count);
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND) {
    return CalcVirParticleGPU(distSq, index, gpu_sigmaSq, gpu_n,
        gpu_epsilon_Cn, gpu_lambdaVDW, sc_sigma_6,
        sc_alpha, sc_power);
  } else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND) {
    return CalcVirShiftGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
        gpu_lambdaVDW, sc_sigma_6, sc_alpha, sc_power);
  } else if(gpu_VDW_Kind == GPU_VDW_EXP6_KIND) {
    return CalcVirExp6GPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_rMin,
        gpu_rMaxSq, gpu_expConst, gpu_lambdaVDW, sc_sigma_6,
        sc_alpha, sc_power);
  } else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini) {
    return CalcVirSwitchMartiniGPU(distSq, index, gpu_sigmaSq, gpu_n,
        gpu_epsilon_Cn, gpu_rCut, gpu_rOn,
        gpu_lambdaVDW, sc_sigma_6, sc_alpha,
        sc_power);
  } else
    return CalcVirSwitchGPU(distSq, index, gpu_sigmaSq, gpu_epsilon_Cn, gpu_n,
        gpu_rCut, gpu_rOn);
}

//ElectroStatic Calculation
//**************************************************************//
__device__ double CalcCoulombVirParticleGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha,
    int index, double *gpu_sigmaSq,
    bool sc_coul, double sc_sigma_6,
    double sc_alpha, uint sc_power,
    double gpu_lambdaCoulomb)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombVirParticleGPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 + dist6;
    double softRsq = pow(softDist6, (double)1.0 / 3.0);
    double correction = distSq / softRsq;
    return gpu_lambdaCoulomb * correction * correction *
      CalcCoulombVirParticleGPU(softRsq, qi_qj, gpu_ewald, gpu_alpha);
  } else {
    return gpu_lambdaCoulomb *
      CalcCoulombVirParticleGPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }
}

__device__ double CalcCoulombVirParticleGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    double dist = sqrt(distSq);
    double result = qi_qj / (distSq * dist);
    return result;
  }
}

__device__ double CalcCoulombVirShiftGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha,
    int index, double *gpu_sigmaSq,
    bool sc_coul, double sc_sigma_6,
    double sc_alpha, uint sc_power,
    double gpu_lambdaCoulomb)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombVirShiftGPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 + dist6;
    double softRsq = pow(softDist6, (double)1.0 / 3.0);
    double correction = distSq / softRsq;
    return gpu_lambdaCoulomb * correction * correction *
      CalcCoulombVirShiftGPU(softRsq, qi_qj, gpu_ewald, gpu_alpha);
  } else {
    return gpu_lambdaCoulomb *
      CalcCoulombVirShiftGPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }
}

__device__ double CalcCoulombVirShiftGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    double dist = sqrt(distSq);
    return qi_qj / (distSq * dist);
  }
}

__device__ double CalcCoulombVirExp6GPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha,
    int index, double *gpu_sigmaSq,
    bool sc_coul, double sc_sigma_6,
    double sc_alpha, uint sc_power,
    double gpu_lambdaCoulomb)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombVirExp6GPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }
  if(sc_coul) {
    double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 + dist6;
    double softRsq = pow(softDist6, (double)1.0 / 3.0);
    double correction = distSq / softRsq;
    return gpu_lambdaCoulomb * correction * correction *
      CalcCoulombVirExp6GPU(softRsq, qi_qj, gpu_ewald, gpu_alpha);
  } else {
    return gpu_lambdaCoulomb *
      CalcCoulombVirExp6GPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }
}

__device__ double CalcCoulombVirExp6GPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = erfc(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    double dist = sqrt(distSq);
    return qi_qj / (distSq * dist);
  }
}

__device__ double CalcCoulombVirSwitchMartiniGPU(double distSq,
    double qi_qj,
    int gpu_ewald,
    double gpu_alpha,
    double gpu_rCut,
    double gpu_diElectric_1,
    int index,
    double *gpu_sigmaSq,
    bool sc_coul,
    double sc_sigma_6,
    double sc_alpha,
    uint sc_power,
    double gpu_lambdaCoulomb)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombVirSwitchMartiniGPU(distSq, qi_qj, gpu_ewald, gpu_alpha,
        gpu_rCut, gpu_diElectric_1);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 + dist6;
    double softRsq = pow(softDist6, 1.0 / 3.0);
    double correction = distSq / softRsq;
    return gpu_lambdaCoulomb * correction * correction *
      CalcCoulombVirSwitchMartiniGPU(softRsq, qi_qj, gpu_ewald, gpu_alpha,
          gpu_rCut, gpu_diElectric_1);
  } else {
    return gpu_lambdaCoulomb *
      CalcCoulombVirSwitchMartiniGPU(distSq, qi_qj, gpu_ewald, gpu_alpha,
          gpu_rCut, gpu_diElectric_1);
  }
}

__device__ double CalcCoulombVirSwitchMartiniGPU(double distSq, double qi_qj,
    int gpu_ewald,
    double gpu_alpha,
    double gpu_rCut,
    double gpu_diElectric_1)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return  qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    // in Martini, the Coulomb switching distance is zero, so we will have
    // sqrt(distSq) - rOnCoul =  sqrt(distSq)
    double dist = sqrt(distSq);
    double rij_ronCoul_2 = distSq;
    double rij_ronCoul_3 = dist * distSq;

    double A1 = 1.0 * (-(1.0 + 4) * gpu_rCut) / (pow(gpu_rCut, 1.0 + 2) *
        pow(gpu_rCut, 2));
    double B1 = -1.0 * (-(1.0 + 3) * gpu_rCut) / (pow(gpu_rCut, 1.0 + 2) *
        pow(gpu_rCut, 3));

    double virCoul = A1 / rij_ronCoul_2 + B1 / rij_ronCoul_3;
    return qi_qj * gpu_diElectric_1 * ( 1.0 / (dist * distSq) + virCoul / dist);
  }
}

__device__ double CalcCoulombVirSwitchGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha,
    double gpu_rCut, int index,
    double *gpu_sigmaSq, bool sc_coul,
    double sc_sigma_6, double sc_alpha,
    uint sc_power,
    double gpu_lambdaCoulomb)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombVirSwitchGPU(distSq, qi_qj, gpu_ewald, gpu_alpha,
        gpu_rCut);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 + dist6;
    double softRsq = pow(softDist6, 1.0 / 3.0);
    double correction = distSq / softRsq;
    return gpu_lambdaCoulomb * correction * correction *
      CalcCoulombVirSwitchGPU(softRsq, qi_qj, gpu_ewald, gpu_alpha, gpu_rCut);
  } else {
    return gpu_lambdaCoulomb *
      CalcCoulombVirSwitchGPU(distSq, qi_qj, gpu_ewald, gpu_alpha, gpu_rCut);
  }
}

__device__ double CalcCoulombVirSwitchGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha,
    double gpu_rCut)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    double rCutSq = gpu_rCut * gpu_rCut;
    double dist = sqrt(distSq);
    double switchVal = distSq / rCutSq - 1.0;
    switchVal *= switchVal;

    double dSwitchVal = 2.0 * (distSq / rCutSq - 1.0) * 2.0 * dist / rCutSq;
    return -1.0 * qi_qj * (dSwitchVal / distSq - switchVal / (distSq * dist));
  }
}

//VDW Calculation
//*****************************************************************//
__device__ double CalcVirParticleGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn,
    double sc_sigma_6,
    double sc_alpha, uint sc_power,
    double gpu_lambdaVDW)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcVirParticleGPU(distSq, index, gpu_sigmaSq, gpu_n,
        gpu_epsilon_Cn);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);
  double correction = distSq / softRsq;
  return gpu_lambdaVDW * correction * correction *
    CalcVirParticleGPU(softRsq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn);
}

__device__ double CalcVirParticleGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0 / distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  return gpu_epsilon_Cn[index] * 6.0 *
    ((gpu_n[index] / 6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirShiftGPU(double distSq, int index, double *gpu_sigmaSq,
    double *gpu_n, double *gpu_epsilon_Cn,
    double sc_sigma_6, double sc_alpha,
    uint sc_power, double gpu_lambdaVDW)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcVirShiftGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);
  double correction = distSq / softRsq;
  return gpu_lambdaVDW * correction * correction *
    CalcVirShiftGPU(softRsq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn);
}

__device__ double CalcVirShiftGPU(double distSq, int index, double *gpu_sigmaSq,
    double *gpu_n, double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0 / distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  return gpu_epsilon_Cn[index] * 6.0 *
    ((gpu_n[index] / 6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirExp6GPU(double distSq, int index, double *gpu_sigmaSq,
    double *gpu_n, double *gpu_rMin,
    double *gpu_rMaxSq, double *gpu_expConst,
    double sc_sigma_6,
    double sc_alpha, uint sc_power,
    double gpu_lambdaVDW)
{
  if(distSq < gpu_rMaxSq[index]) {
    return num::BIGNUM;
  }
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcVirExp6GPU(distSq, index, gpu_n, gpu_rMin, gpu_expConst);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);
  double correction = distSq / softRsq;
  return gpu_lambdaVDW * correction * correction *
    CalcVirExp6GPU(softRsq, index, gpu_n, gpu_rMin, gpu_expConst);
}

__device__ double CalcVirExp6GPU(double distSq, int index, double *gpu_n,
    double *gpu_rMin, double *gpu_expConst)
{
  double dist = sqrt(distSq);
  double rRat = gpu_rMin[index] / dist;
  double rRat2 = rRat * rRat;
  double attract = rRat2 * rRat2 * rRat2;

  uint alpha_ij = gpu_n[index];
  double repulse = (dist / gpu_rMin[index]) * exp(alpha_ij *
      (1.0 - dist / gpu_rMin[index]));
  return 6.0 * gpu_expConst[index] * (repulse - attract) / distSq;
}

__device__ double CalcVirSwitchMartiniGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn,
    double gpu_rCut, double gpu_rOn,
    double sc_sigma_6, double sc_alpha,
    uint sc_power,
    double gpu_lambdaVDW)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcVirSwitchMartiniGPU(distSq, index, gpu_sigmaSq, gpu_n,
        gpu_epsilon_Cn, gpu_rCut, gpu_rOn);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);
  double correction = distSq / softRsq;
  return gpu_lambdaVDW * correction * correction *
    CalcVirSwitchMartiniGPU(softRsq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
        gpu_rCut, gpu_rOn);
}

__device__ double CalcVirSwitchMartiniGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn,
    double gpu_rCut, double gpu_rOn)
{
  double r_1 = 1.0 / sqrt(distSq);
  double r_8 = pow(r_1, 8);
  double r_n2 = pow(r_1, gpu_n[index] + 2);

  double rij_ron = sqrt(distSq) - gpu_rOn;
  double rij_ron_2 = rij_ron * rij_ron;
  double rij_ron_3 = rij_ron_2 * rij_ron;

  double pn = gpu_n[index];
  double An = pn * ((pn + 1) * gpu_rOn - (pn + 4) * gpu_rCut) /
    (pow(gpu_rCut, pn + 2) * pow(gpu_rCut - gpu_rOn, 2));
  double Bn = -pn * ((pn + 1) * gpu_rOn - (pn + 3) * gpu_rCut) /
    (pow(gpu_rCut, pn + 2) * pow(gpu_rCut - gpu_rOn, 3));

  double sig6 = pow(gpu_sigmaSq[index], 3);
  double sign = pow(gpu_sigmaSq[index], pn / 2);

  double A6 = 6.0 * ((6.0 + 1) * gpu_rOn - (6.0 + 4) * gpu_rCut) /
    (pow(gpu_rCut, 6.0 + 2) * pow(gpu_rCut - gpu_rOn, 2));
  double B6 = -6.0 * ((6.0 + 1) * gpu_rOn - (6.0 + 3) * gpu_rCut) /
    (pow(gpu_rCut, 6.0 + 2) * pow(gpu_rCut - gpu_rOn, 3));

  double dshifttempRep = An * rij_ron_2 + Bn * rij_ron_3;
  double dshifttempAtt = A6 * rij_ron_2 + B6 * rij_ron_3;

  const double dshiftRep = ( distSq > gpu_rOn * gpu_rOn ?
      dshifttempRep * r_1 : 0);
  const double dshiftAtt = ( distSq > gpu_rOn * gpu_rOn ?
      dshifttempAtt * r_1 : 0);
  double Wij = gpu_epsilon_Cn[index] * (sign * (pn * r_n2 + dshiftRep) -
      sig6 * (6.0 * r_8 + dshiftAtt));
  return Wij;
}

__device__ double CalcVirSwitchGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_epsilon_Cn,
    double *gpu_n, double gpu_rCut,
    double gpu_rOn,
    double sc_sigma_6, double sc_alpha,
    uint sc_power,
    double gpu_lambdaVDW)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcVirSwitchGPU(distSq, index, gpu_sigmaSq, gpu_epsilon_Cn, gpu_n,
        gpu_rCut, gpu_rOn);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);
  double correction = distSq / softRsq;
  return gpu_lambdaVDW * correction * correction *
    CalcVirSwitchGPU(softRsq, index, gpu_sigmaSq, gpu_epsilon_Cn, gpu_n,
        gpu_rCut, gpu_rOn);
}

__device__ double CalcVirSwitchGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_epsilon_Cn,
    double *gpu_n, double gpu_rCut,
    double gpu_rOn)
{
  double rCutSq = gpu_rCut * gpu_rCut;
  double rCutSq_rijSq = rCutSq - distSq;
  double rCutSq_rijSq_Sq = rCutSq_rijSq * rCutSq_rijSq;
  double rOnSq = gpu_rOn * gpu_rOn;

  double rNeg2 = 1.0 / distSq;
  double rRat2 = rNeg2 * gpu_sigmaSq[index];
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  double factor1 = rCutSq - 3 * rOnSq;
  double factor2 = pow((rCutSq - rOnSq), -3);

  double fE = rCutSq_rijSq_Sq * factor2 * (factor1 + 2 * distSq);
  double fW = 12.0 * factor2 * rCutSq_rijSq * (rOnSq - distSq);

  const double factE = ( distSq > rOnSq ? fE : 1.0);
  const double factW = ( distSq > rOnSq ? fW : 0.0);

  double Wij = gpu_epsilon_Cn[index] * 6.0 *
    ((gpu_n[index] / 6.0) * repulse - attract) * rNeg2;
  double Eij = gpu_epsilon_Cn[index] * (repulse - attract);

  return (Wij * factE - Eij * factW);
}

#endif
